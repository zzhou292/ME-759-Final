#include "hip/hip_runtime.h"
#include "chrono_vehicle/cuda/hello.cuh"
#include <stdio.h>
#include <stdlib.h>

__global__ void cuda_hello() {
  printf("Hello from Thread %d in block %d\n", threadIdx.x, blockIdx.x);
}

void wrapper() {
  cuda_hello<<<1, 1>>>();
  hipDeviceSynchronize();
}
